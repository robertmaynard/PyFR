#include "hip/hip_runtime.h"
#include "PyFRData.h"

#include <algorithm>
#include <cassert>
#include <iostream>
#include <iterator>
#include <limits>
#include <sstream>
#include <stdexcept>

#include <vtkDataObjectTypes.h>
#include <vtkObjectFactory.h>

#include <vtkm/TopologyElementTag.h>
#include <vtkm/cont/ArrayHandle.h>
#include <vtkm/cont/ArrayHandleCast.h>
#include <vtkm/cont/CoordinateSystem.h>
#include <vtkm/cont/DeviceAdapter.h>
#include <vtkm/cont/DataSet.h>
#include <vtkm/cont/Field.h>
#include <vtkm/cont/internal/DeviceAdapterTag.h>
#include <vtkm/cont/cuda/ArrayHandleCuda.h>
#include <vtkm/cont/cuda/internal/DeviceAdapterTagCuda.h>

namespace
{
  typedef vtkm::Id Id;
  typedef vtkm::Id3 Id3;
  typedef vtkm::Vec<double,3> Double3;
  typedef vtkm::cont::ArrayHandle<Double3> Double3ArrayHandle;
  struct GridData
  {
    Id3 Dimension;
    Double3 Origin;
    Double3 Spacing;
  };

  GridData ComputeGridDimensions(Double3ArrayHandle&);
};

//------------------------------------------------------------------------------
vtkStandardNewMacro(PyFRData);

//------------------------------------------------------------------------------
PyFRData::PyFRData() : catalystData(NULL)
{

}

//------------------------------------------------------------------------------
PyFRData::~PyFRData()
{
}

//------------------------------------------------------------------------------
void PyFRData::Init(vtkIdType datasettypeid, void* data)
{
  this->dataSetTypeId = datasettypeid;
  assert(this->dataSetTypeId = vtkDataObjectTypes::GetTypeIdFromClassName("vtkStructuredGrid"));

  this->catalystData = static_cast<struct CatalystData*>(data);

  MeshDataForCellType* meshData = &(this->catalystData->meshData[0]);
  SolutionDataForCellType* solutionData= &(this->catalystData->solutionData[0]);

  const size_t xyz_len = (meshData->nCells*
                          meshData->nVerticesPerCell);
  const double* vbuf = meshData->vertices;

  vtkm::cont::ArrayHandle<vtkm::Vec<double,3> > vertices =
    vtkm::cont::ArrayHandle<vtkm::Vec<double,3>,vtkm::cont::StorageTagBasic>(vtkm::cont::internal::Storage<vtkm::Vec<double,3>,vtkm::cont::StorageTagBasic>(reinterpret_cast<const vtkm::Vec<double,3>*>(vbuf),xyz_len));
  GridData gridData = ComputeGridDimensions(vertices);
  // std::cout<<"Grid data:"<<std::endl;
  // std::cout<<"  dimension: "<<gridData.Dimension<<std::endl;
  // std::cout<<"  origin:    "<<gridData.Origin<<std::endl;
  // std::cout<<"  spacing:   "<<gridData.Spacing<<std::endl;

  for (int i=0;i<3;i++)
    this->cellDimension[i] = gridData.Dimension[i] - 1;

  vtkm::cont::CellSetStructured<3> cset("cells");
  cset.SetPointDimensions(gridData.Dimension);

  vtkm::cont::cuda::ArrayHandleCuda<double>::type solution =
    vtkm::cont::cuda::make_ArrayHandle(
      static_cast<double*>(solutionData->solution),
      meshData->nCells*meshData->nVerticesPerCell);

  enum ElemType { CONSTANT=0, LINEAR=1, QUADRATIC=2 };
  vtkm::cont::Field rho("rho",LINEAR,vtkm::cont::Field::ASSOC_POINTS,solution);

  this->dataSet.AddCoordinateSystem(
    vtkm::cont::CoordinateSystem("coordinates",
                                 1,
                                 gridData.Dimension,
                                 gridData.Origin,
                                 gridData.Spacing));
  this->dataSet.AddField(rho);
  this->dataSet.AddCellSet(cset);
}

//------------------------------------------------------------------------------
void PyFRData::Update()
{
}

//----------------------------------------------------------------------------
namespace
{
GridData ComputeGridDimensions(Double3ArrayHandle& ptsArray)
{
  const double epsilon = 1.e-6;

  Double3ArrayHandle::PortalConstControl points =
    ptsArray.GetPortalConstControl();

  Id3 xyz;
  xyz[0] = xyz[1] = xyz[2] = 0;
  Double3 sentinel = points.Get(0);
  Double3 spacing;
  for (Id i=0;i<3;i++) spacing[i] = std::numeric_limits<double>::max();
  Double3 point;
  Id counter = 0;
  Id increment = 1;
  Id3 indexing;
  indexing[0] = indexing[1] = indexing[2] = -1;

  while (counter < ptsArray.GetNumberOfValues())
    {
    counter += increment;
    point = points.Get(counter);

    for (Id i=0;i<3;i++)
      {
      if (fabs(point[i]-sentinel[i]) > epsilon &&
          fabs(point[i]-sentinel[i]) < spacing[i])
        spacing[i] = fabs(point[i]-sentinel[i]);
      }

    if (indexing[0] == -1)
      {
      for (Id i=0;i<3;i++)
        {
        if (fabs(point[i]-sentinel[i]) > epsilon)
          {
          for (Id j=0;j<3;j++)
            indexing[j] = (i+j)%3;
          }
        }
      }

    if (xyz[indexing[0]] == 0)
      {
      if (fabs(point[indexing[0]]-sentinel[indexing[0]]) < epsilon)
        {
        xyz[indexing[0]] = increment = counter;
        point = points.Get(counter+increment);
        for (Id i=0;i<3;i++)
          {
          if (i == indexing[0])
            continue;
          if (fabs(point[i]-sentinel[i])>epsilon)
            {
            indexing[1] = i;
            for (Id j=0;j<3;j++)
              {
              if (j != indexing[0] && j != indexing[1])
                indexing[2] = j;
              }
            }
          }
        point = points.Get(counter);
        }
      }

    if (xyz[indexing[0]] != 0 && xyz[indexing[1]] == 0)
      {
      if (fabs(point[indexing[1]]-sentinel[indexing[1]]) < epsilon)
        {
        xyz[indexing[1]] = counter/xyz[indexing[0]];
        increment = counter;
        }
      }

    if (xyz[indexing[1]] != 0 && xyz[indexing[2]] == 0)
      {
      if (fabs(point[indexing[2]]-sentinel[indexing[2]]) < epsilon)
        {
        xyz[indexing[2]] = counter/xyz[indexing[0]]/xyz[indexing[1]];
        break;
        }
      }
    }
  if (xyz[indexing[2]] == 0)
    xyz[indexing[2]] = ptsArray.GetNumberOfValues()/xyz[indexing[0]]/xyz[indexing[1]];

  GridData gridData;
  gridData.Dimension = xyz;
  gridData.Origin = points.Get(xyz[0]*xyz[1]*xyz[2]*3); // ???
  gridData.Spacing = spacing;

  return gridData;
}
}
