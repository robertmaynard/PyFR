#include "vtkCPVTKmPipeline.h"

#include <sstream>

#include <hip/hip_runtime.h>

#include "PyFRData.h"

#include <vtkCommunicator.h>
#include <vtkCompleteArrays.h>
#include <vtkCPDataDescription.h>
#include <vtkCPInputDataDescription.h>
#include <vtkDataArray.h>
#include <vtkNew.h>
#include <vtkObjectFactory.h>
#include <vtkPointData.h>

#include <vtkm/cont/ArrayHandle.h>
#include <vtkm/cont/DeviceAdapter.h>
#include <vtkm/cont/Field.h>
#include <vtkm/cont/DataSet.h>
#include <vtkm/cont/internal/DeviceAdapterTag.h>
#include <vtkm/cont/cuda/ArrayHandleCuda.h>
#include <vtkm/cont/cuda/internal/DeviceAdapterTagCuda.h>
#include <vtkm/worklet/DispatcherMapField.h>
#include <vtkm/worklet/ExternalFaces.h>
#include <vtkm/worklet/WorkletMapField.h>

vtkStandardNewMacro(vtkCPVTKmPipeline);

//----------------------------------------------------------------------------
vtkCPVTKmPipeline::vtkCPVTKmPipeline()
{
}

//----------------------------------------------------------------------------
vtkCPVTKmPipeline::~vtkCPVTKmPipeline()
{
}

//----------------------------------------------------------------------------
void vtkCPVTKmPipeline::Initialize(char* fileName)
{
  this->fileName = std::string(fileName);
}

//----------------------------------------------------------------------------
int vtkCPVTKmPipeline::RequestDataDescription(
  vtkCPDataDescription* dataDescription)
{
  if(!dataDescription)
    {
    vtkWarningMacro("dataDescription is NULL.");
    return 0;
    }

  if(this->fileName.empty())
    {
    vtkWarningMacro("No output file name given to output results to.");
    return 0;
    }

  dataDescription->GetInputDescriptionByName("input")->AllFieldsOn();
  dataDescription->GetInputDescriptionByName("input")->GenerateMeshOn();
  return 1;
}

//----------------------------------------------------------------------------
int vtkCPVTKmPipeline::CoProcess(vtkCPDataDescription* dataDescription)
{
  if(!dataDescription)
    {
    vtkWarningMacro("DataDescription is NULL");
    return 0;
    }
  PyFRData* pyfrData =
    PyFRData::SafeDownCast(dataDescription->
                           GetInputDescriptionByName("input")->GetGrid());
  if(pyfrData == NULL)
    {
    vtkWarningMacro("DataDescription is missing input PyFR data.");
    return 0;
    }
  if(this->RequestDataDescription(dataDescription) == 0)
    {
    return 1;
    }

  vtkm::cont::DataSet& dataSet = pyfrData->GetDataSet();

  vtkm::cont::CellSetExplicit<>& cset =
    dataSet.GetCellSet(0).CastTo<vtkm::cont::CellSetExplicit<> >();
  vtkm::cont::ArrayHandle<vtkm::Id> shapes = cset.GetShapesArray(
    vtkm::TopologyElementTagPoint(),vtkm::TopologyElementTagCell());
  vtkm::cont::ArrayHandle<vtkm::Id> nindices = cset.GetNumIndicesArray(
    vtkm::TopologyElementTagPoint(),vtkm::TopologyElementTagCell());
  vtkm::cont::ArrayHandle<vtkm::Id> conn = cset.GetConnectivityArray(
    vtkm::TopologyElementTagPoint(),vtkm::TopologyElementTagCell());
  vtkm::cont::ArrayHandle<vtkm::Id> shapes_out;
  vtkm::cont::ArrayHandle<vtkm::Id> nindices_out;
  vtkm::cont::ArrayHandle<vtkm::Id> conn_out;

  vtkm::worklet::ExternalFaces<VTKM_DEFAULT_DEVICE_ADAPTER_TAG>().run(
    shapes, nindices, conn, shapes_out, nindices_out, conn_out
  );
  vtkm::cont::DataSet outDataSet;
  for(size_t i=0; i < (size_t)dataSet.GetNumberOfCoordinateSystems(); ++i)
    {
    outDataSet.AddCoordinateSystem(dataSet.GetCoordinateSystem(i));
    }

  std::cout << shapes.GetNumberOfValues() << " input elements, "
            << shapes_out.GetNumberOfValues() << " output elements.\n";

/*
  vm::CellSetExplicit<> outcset("cells", shapes_out.GetNumberOfValues());
  outcset.Fill(shapes_out, nindices_out, conn_out);
  outds.AddCellSet(outcset);

  vtkUnstructuredGrid* grid = vtkUnstructuredGrid::SafeDownCast(
    dataDescription->GetInputDescriptionByName("input")->GetGrid());
  if(grid == NULL)
    {
    vtkWarningMacro("DataDescription is missing input unstructured grid.");
    return 0;
    }
  if(this->RequestDataDescription(dataDescription) == 0)
    {
    return 1;
    }
*/

/*
  vtkNew<vtkPVTrivialProducer> producer;
  producer->SetOutput(grid);

  vtkNew<vtkPVArrayCalculator> calculator;
  calculator->SetInputConnection(producer->GetOutputPort());
  calculator->SetAttributeMode(1);
  calculator->SetResultArrayName("velocity magnitude");
  calculator->SetFunction("mag(velocity)");

  // update now so that we can get the global data bounds of
  // the velocity magnitude for thresholding
  calculator->Update();
  double range[2];
  vtkUnstructuredGrid::SafeDownCast(calculator->GetOutput())->GetPointData()
    ->GetArray("velocity magnitude")->GetRange(range, 0);
  double globalRange[2];
  vtkMultiProcessController::GetGlobalController()->AllReduce(
    range+1, globalRange+1, 1, vtkCommunicator::MAX_OP);

  vtkNew<vtkThreshold> threshold;
  threshold->SetInputConnection(calculator->GetOutputPort());
  threshold->SetInputArrayToProcess(
    0, 0, 0, "vtkDataObject::FIELD_ASSOCIATION_POINTS", "velocity magnitude");
  threshold->ThresholdBetween(0.9*globalRange[1], globalRange[1]);

  // If process 0 doesn't have any points or cells, the writer may
  // have problems in parallel so we use completeArrays to fill in
  // the missing information.
  vtkNew<vtkCompleteArrays> completeArrays;
  completeArrays->SetInputConnection(threshold->GetOutputPort());

  vtkNew<vtkXMLPUnstructuredGridWriter> writer;
  writer->SetInputConnection(completeArrays->GetOutputPort());
  std::ostringstream o;
  o << dataDescription->GetTimeStep();
  std::string name = this->fileName + o.str() + ".pvtu";
  writer->SetfileName(name.c_str());
  writer->Update();
*/
  return 1;
}

//----------------------------------------------------------------------------
void vtkCPVTKmPipeline::PrintSelf(ostream& os, vtkIndent indent)
{
  this->Superclass::PrintSelf(os, indent);
  os << indent << "fileName: " << this->fileName << "\n";
}
