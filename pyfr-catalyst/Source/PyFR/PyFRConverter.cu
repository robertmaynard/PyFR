#include <cmath>
#include <iomanip>
#include <string>
#include <sstream>

#include "PyFRConverter.h"

#include <vtkAppendPolyData.h>
#include <vtkCellArray.h>
#include <vtkCellType.h>
#include <vtkCommand.h>
#include <vtkCommunicator.h>
#include <vtkCompleteArrays.h>
#include <vtkCPDataDescription.h>
#include <vtkCPInputDataDescription.h>
#include <vtkDataArray.h>
#include <vtkDoubleArray.h>
#include "vtkErrorCode.h"
#include "vtkExecutive.h"
#include <vtkFloatArray.h>
#include <vtkHexahedron.h>
#include <vtkIdTypeArray.h>
#include <vtkNew.h>
#include <vtkObjectFactory.h>
#include <vtkPointData.h>
#include <vtkPoints.h>
#include <vtkPolyData.h>
#include <vtkSmartPointer.h>
#include <vtkUnstructuredGrid.h>

#include <vtkm/cont/ArrayHandleCast.h>
#include <vtkm/cont/DeviceAdapter.h>
#include <vtkm/cont/DeviceAdapterAlgorithm.h>
#include <vtkm/cont/DeviceAdapterSerial.h>
#include <vtkm/cont/DynamicArrayHandle.h>
#include <vtkm/cont/cuda/ArrayHandleCuda.h>
#include <vtkm/cont/cuda/DeviceAdapterCuda.h>

#include "ArrayHandleExposed.h"
#include "PyFRData.h"
#include "PyFRContour.h"
#include "PyFRContourData.h"

template <typename fptype>
struct ArrayChoice;

template <>
struct ArrayChoice<float>
{
  typedef vtkFloatArray type;
};

template <>
struct ArrayChoice<double>
{
  typedef vtkDoubleArray type;
};

//----------------------------------------------------------------------------
PyFRConverter::PyFRConverter()
{
}

//----------------------------------------------------------------------------
PyFRConverter::~PyFRConverter()
{
}

//----------------------------------------------------------------------------
void PyFRConverter::operator ()(const PyFRData* pyfrData,vtkUnstructuredGrid* grid) const
{
  const vtkm::cont::DataSet& dataSet = pyfrData->GetDataSet();

  namespace vtkmc = vtkm::cont;
  typedef vtkmc::ArrayHandleExposed<FPType> ScalarDataArrayHandleExposed;
  typedef vtkmc::ArrayHandleExposed<vtkm::Vec<FPType,3> >
    Vec3ArrayHandleExposed;
  typedef ::vtkm::cont::DeviceAdapterTagCuda CudaTag;

  Vec3ArrayHandleExposed vertices;
    {
    PyFRData::Vec3ArrayHandle tmp = dataSet.GetCoordinateSystem().GetData()
      .CastToArrayHandle(PyFRData::Vec3ArrayHandle::ValueType(),
                         PyFRData::Vec3ArrayHandle::StorageTag());
    vtkm::cont::DeviceAdapterAlgorithm<CudaTag>().
      Copy(tmp,vertices);
    }

  vtkSmartPointer<ArrayChoice<FPType>::type> pointData =
    vtkSmartPointer<ArrayChoice<FPType>::type>::New();

  vtkIdType nVerts = vertices.GetNumberOfValues();
  FPType* vertsArray = reinterpret_cast<FPType*>(vertices.Storage().StealArray());
  pointData->SetArray(vertsArray, nVerts*3,
                      0, // give VTK control of the data
                      0);// delete using "free"
  pointData->SetNumberOfComponents(3);

  vtkSmartPointer<vtkPoints> points = vtkSmartPointer<vtkPoints>::New();
  points->SetData(pointData);

  std::string fieldName[5] = {"density","velocity_u","velocity_v","velocity_w",
"pressure"};
  vtkSmartPointer<ArrayChoice<FPType>::type> solutionData[5];
  for (unsigned i=0;i<5;i++)
    {
    vtkmc::Field solution = dataSet.GetField(fieldName[i]);
    PyFRData::ScalarDataArrayHandle solutionArray = solution.GetData()
      .CastToArrayHandle(PyFRData::ScalarDataArrayHandle::ValueType(),
                         PyFRData::ScalarDataArrayHandle::StorageTag());
    ScalarDataArrayHandleExposed solutionArrayHost;
    vtkm::cont::DeviceAdapterAlgorithm<CudaTag>().
      Copy(solutionArray, solutionArrayHost);

    solutionData[i] = vtkSmartPointer<ArrayChoice<FPType>::type>::New();
    vtkIdType nSolution = solutionArrayHost.GetNumberOfValues();
    FPType* solutionArr = solutionArrayHost.Storage().StealArray();
    solutionData[i]->SetArray(solutionArr, nSolution,
                           0, // give VTK control of the data
                           0);// delete using "free"
    solutionData[i]->SetNumberOfComponents(1);
    solutionData[i]->SetName(fieldName[i].c_str());
    }

  PyFRData::CellSet cellSet = dataSet.GetCellSet().CastTo(PyFRData::CellSet());

    vtkm::cont::ArrayHandle<vtkm::Id> connectivity =
      cellSet.GetConnectivityArray(vtkm::TopologyElementTagPoint(),
                                   vtkm::TopologyElementTagCell());
    vtkm::cont::ArrayHandle<vtkm::Id>::PortalConstControl portal =
      connectivity.GetPortalConstControl();

  grid->Allocate(connectivity.GetNumberOfValues()/8);
  grid->SetPoints(points);
  for (unsigned i=0;i<5;i++)
    grid->GetPointData()->AddArray(solutionData[i]);
  vtkIdType counter = 0;
  while (counter < connectivity.GetNumberOfValues())
    {
    vtkSmartPointer<vtkHexahedron> hex = vtkSmartPointer<vtkHexahedron>::New();
    for (vtkIdType j=0;j<8;j++)
      hex->GetPointIds()->SetId(j,portal.Get(counter++));
    grid->InsertNextCell(hex->GetCellType(),hex->GetPointIds());
    }
}

//----------------------------------------------------------------------------
void PyFRConverter::operator ()(const PyFRContour& contour,vtkPolyData* polydata) const
{
  typedef ::vtkm::cont::DeviceAdapterTagCuda CudaTag;

  typedef vtkm::cont::ArrayHandleExposed<vtkm::Vec<FPType,3> > Vec3ArrayHandle;

  if (contour.GetVertices().GetNumberOfValues() == 0)
    {
//    polydata->Reset();
    return;
    }

  PyFRContour::Vec3ArrayHandle verts_out;
  vtkm::cont::DeviceAdapterAlgorithm<CudaTag>().
    Copy(contour.GetVertices(),verts_out);

  vtkSmartPointer<ArrayChoice<FPType>::type> pointData =
    vtkSmartPointer<ArrayChoice<FPType>::type>::New();

  vtkIdType nVerts = verts_out.GetNumberOfValues();
  FPType* vertsArray = reinterpret_cast<FPType*>(verts_out.Storage().StealArray());
  pointData->SetArray(vertsArray, nVerts*3,
                      0, // give VTK control of the data
                      0);// delete using "free"
  pointData->SetNumberOfComponents(3);

  vtkSmartPointer<vtkPoints> points = vtkSmartPointer<vtkPoints>::New();
  points->SetData(pointData);

  PyFRContour::Vec3ArrayHandle normals_out;
  vtkm::cont::DeviceAdapterAlgorithm<CudaTag>().
    Copy(contour.GetNormals(),normals_out);

  vtkSmartPointer<ArrayChoice<FPType>::type> normalsData =
    vtkSmartPointer<ArrayChoice<FPType>::type>::New();

  vtkIdType nNormals = normals_out.GetNumberOfValues();
  FPType* normalsArray = reinterpret_cast<FPType*>(normals_out.Storage().StealArray());
  normalsData->SetArray(normalsArray, nNormals*3,
                        0, // give VTK control of the data
                        0);// delete using "free"
  normalsData->SetNumberOfComponents(3);

  vtkSmartPointer<vtkCellArray> polys =
        vtkSmartPointer<vtkCellArray>::New();
  vtkIdType indices[3];
  for (vtkIdType i=0;i<points->GetNumberOfPoints();i+=3)
    {
    for (vtkIdType j=0;j<3;j++)
      indices[j] = i+j;
    polys->InsertNextCell(3,indices);
    }

  polydata->SetPoints(points);
  polydata->SetPolys(polys);
  polydata->GetPointData()->SetNormals(normalsData);

  std::string fields[5] = {"density","pressure","velocity_u","velocity_v","velocity_w"};
  for (unsigned i=0;i<5;i++)
    {
    PyFRContour::ScalarDataArrayHandle scalarsOut = contour.GetScalarData(fields[i]);
    PyFRContour::ScalarDataArrayHandle scalarsOutHost;
    vtkm::cont::DeviceAdapterAlgorithm<CudaTag>().
      Copy(scalarsOut,scalarsOutHost);

    vtkSmartPointer<ArrayChoice<FPType>::type> solutionData =
      vtkSmartPointer<ArrayChoice<FPType>::type>::New();
    vtkIdType nSolution = scalarsOutHost.GetNumberOfValues();
    FPType* solutionArray = scalarsOutHost.Storage().StealArray();
    solutionData->SetArray(solutionArray, nSolution,
                           0, // give VTK control of the data
                           0);// delete using "free"
    solutionData->SetNumberOfComponents(1);
    solutionData->SetName(fields[i].c_str());

    polydata->GetPointData()->AddArray(solutionData);
    }
}

//----------------------------------------------------------------------------
void PyFRConverter::operator ()(const PyFRContourData* pyfrContourData,vtkPolyData* outPolyData) const
{
  vtkSmartPointer<vtkAppendPolyData> appendFilter =
    vtkSmartPointer<vtkAppendPolyData>::New();
  appendFilter->SetOutput(outPolyData);

  std::vector<vtkPolyData*> polyData(pyfrContourData->GetNumberOfContours(),NULL);

  for (unsigned i=0;i<pyfrContourData->GetNumberOfContours();i++)
    {
    polyData[i] = vtkPolyData::New();
    this->operator()(pyfrContourData->GetContour(i),polyData[i]);
    appendFilter->AddInputData(polyData[i]);
    }

  appendFilter->Update();

  for (unsigned i=0;i<pyfrContourData->GetNumberOfContours();i++)
    polyData[i]->Delete();
}