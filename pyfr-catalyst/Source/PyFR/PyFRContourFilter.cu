#include "PyFRContourFilter.h"

#include "CrinkleClip.h"
#include "PyFRData.h"
#include "PyFRContourData.h"

//----------------------------------------------------------------------------
PyFRContourFilter::PyFRContourFilter() : ContourField(0)
{
}

//----------------------------------------------------------------------------
PyFRContourFilter::~PyFRContourFilter()
{
}

//----------------------------------------------------------------------------
void PyFRContourFilter::operator()(PyFRData* input,
                                   PyFRContourData* output)
{
  typedef std::vector<vtkm::cont::ArrayHandle<vtkm::Vec<FPType,3> > >
    Vec3HandleVec;
  typedef std::vector<FPType> DataVec;
  typedef vtkm::worklet::CrinkleClipTraits<typename PyFRData::CellSet>::CellSet
    CellSet;

  const vtkm::cont::DataSet& dataSet = input->GetDataSet();

  vtkm::cont::Field contourField =
    dataSet.GetField(PyFRData::FieldName(this->ContourField));
  PyFRData::ScalarDataArrayHandle contourArray = contourField.GetData()
    .CastToArrayHandle(PyFRData::ScalarDataArrayHandle::ValueType(),
                       PyFRData::ScalarDataArrayHandle::StorageTag());

  DataVec dataVec;
  Vec3HandleVec verticesVec;
  Vec3HandleVec normalsVec;
  output->SetNumberOfContours(this->ContourValues.size());
  for (unsigned i=0;i<output->GetNumberOfContours();i++)
    {
    dataVec.push_back(this->ContourValues[i]);
    verticesVec.push_back(output->GetContour(i).GetVertices());
    normalsVec.push_back(output->GetContour(i).GetNormals());
    }

  isosurfaceFilter.Run(dataVec,
                       dataSet.GetCellSet().CastTo(CellSet()),
                       dataSet.GetCoordinateSystem(),
                       contourArray,
                       verticesVec,
                       normalsVec);
}

//----------------------------------------------------------------------------
void PyFRContourFilter::MapFieldOntoIsosurfaces(int field,
                                                PyFRData* input,
                                                PyFRContourData* output)
{
  typedef std::vector<vtkm::cont::ArrayHandle<FPType> > ScalarDataHandleVec;

  const vtkm::cont::DataSet& dataSet = input->GetDataSet();

  ScalarDataHandleVec scalarDataHandleVec;
  for (unsigned j=0;j<output->GetNumberOfContours();j++)
    {
    output->GetContour(j).SetScalarDataType(field);
    PyFRContour::ScalarDataArrayHandle scalars_out =
      output->GetContour(j).GetScalarData();
    scalarDataHandleVec.push_back(scalars_out);
    }

  vtkm::cont::Field projectedField =
    dataSet.GetField(PyFRData::FieldName(field));

  PyFRData::ScalarDataArrayHandle projectedArray = projectedField.GetData()
    .CastToArrayHandle(PyFRData::ScalarDataArrayHandle::ValueType(),
                       PyFRData::ScalarDataArrayHandle::StorageTag());

  isosurfaceFilter.MapFieldOntoIsosurfaces(projectedArray,
                                           scalarDataHandleVec);
}
