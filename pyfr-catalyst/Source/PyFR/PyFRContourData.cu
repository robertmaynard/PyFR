#include "hip/hip_runtime.h"
#include "PyFRContourData.h"

#include "Bounds.h"

#include <vtkm/Math.h>
#include <vtkm/Pair.h>
#include <vtkm/Types.h>
#include <vtkm/VectorAnalysis.h>

#include <vtkm/cont/cuda/DeviceAdapterCuda.h>
#include <vtkm/cont/DeviceAdapter.h>
#include <vtkm/cont/ArrayHandleTransform.h>

//----------------------------------------------------------------------------
void PyFRContourData::SetNumberOfContours(unsigned nContours)
{
  // NB: Cannot call resize to increase the lengths of vectors of array
  // handles (or classes containing them)! You will end up with a vector of
  // smart pointers to the same array instance. A specialization of
  // std::allocator<> for array handles should be created.
  for (unsigned i=this->Contours.size();i<nContours;i++)
    this->Contours.push_back(PyFRContour());
  this->Contours.resize(nContours);
}

//----------------------------------------------------------------------------
unsigned PyFRContourData::GetContourSize(int contour) const
{
  return this->GetContour(contour).GetVertices().GetNumberOfValues();
}

//----------------------------------------------------------------------------
void PyFRContourData::ComputeContourBounds(int contour,FPType* bounds) const
{
  typedef ::vtkm::cont::DeviceAdapterTagCuda CudaTag;
  typedef vtkm::cont::DeviceAdapterAlgorithm<CudaTag> Algorithm;
  typedef vtkm::Vec<vtkm::Float64, 3> ResultType;
  typedef vtkm::Pair<ResultType, ResultType> MinMaxPairType;
  typedef PyFRContour::Vec3ArrayHandle ArrayHandleType;

  MinMaxPairType initialValue =
    make_Pair(ResultType(vtkm::Infinity64()),
              ResultType(vtkm::NegativeInfinity64()));

  vtkm::cont::ArrayHandleTransform<MinMaxPairType, ArrayHandleType,
    internal::InputToOutputTypeTransform<3> > input(this->GetContour(contour)
                                                    .GetVertices());

  MinMaxPairType result = Algorithm::Reduce(input, initialValue,
                                            internal::MinMax<3>());

  for (unsigned i=0;i<3;i++)
    {
    bounds[2*i] = result.first[i];
    bounds[2*i+1] = result.second[i];
    }
}

//----------------------------------------------------------------------------
void PyFRContourData::ComputeBounds(FPType* bounds) const
{
  for (unsigned i=0;i<3;i++)
    {
    bounds[2*i] = std::numeric_limits<FPType>::max();
    bounds[2*i+1] = std::numeric_limits<FPType>::min();
    }
  for (unsigned i=0;i<this->GetNumberOfContours();i++)
    {
    FPType b[6];
    this->ComputeContourBounds(i,b);
    for (unsigned j=0;j<3;j++)
      {
      int jj = 2*j;
      bounds[jj] = (bounds[jj] < b[jj] ? bounds[jj] : b[jj]);
      jj++;
      bounds[jj] = (bounds[jj] > b[jj] ? bounds[jj] : b[jj]);
      }
    }
}