#include "hip/hip_runtime.h"
#include "PyFRContourData.h"

#include "Bounds.h"

#include <vtkm/Math.h>
#include <vtkm/Pair.h>
#include <vtkm/Types.h>
#include <vtkm/VectorAnalysis.h>

#include <vtkm/cont/cuda/DeviceAdapterCuda.h>
#include <vtkm/cont/DeviceAdapter.h>
#include <vtkm/cont/ArrayHandleTransform.h>

#include <vtkm/cont/ArrayHandleCast.h>
#include <vtkm/opengl/TransferToOpenGL.h>
#include <vtkm/opengl/cuda/internal/TransferToOpenGL.h>

//----------------------------------------------------------------------------
void PyFRContourData::SetNumberOfContours(unsigned nContours)
{
  // NB: Cannot call resize to increase the lengths of vectors of array
  // handles (or classes containing them)! You will end up with a vector of
  // smart pointers to the same array instance. A specialization of
  // std::allocator<> for array handles should be created.
  for (unsigned i=this->Contours.size();i<nContours;i++)
    this->Contours.push_back(PyFRContour());
  this->Contours.resize(nContours);
}

//----------------------------------------------------------------------------
unsigned PyFRContourData::GetContourSize(int contour) const
{
  return this->GetContour(contour).GetVertices().GetNumberOfValues();
}

//----------------------------------------------------------------------------
void PyFRContourData::ComputeContourBounds(int contour,FPType* bounds) const
{
  typedef ::vtkm::cont::DeviceAdapterTagCuda CudaTag;
  typedef vtkm::cont::DeviceAdapterAlgorithm<CudaTag> Algorithm;
  typedef vtkm::Vec<vtkm::Float64, 3> ResultType;
  typedef vtkm::Pair<ResultType, ResultType> MinMaxPairType;
  typedef PyFRContour::Vec3ArrayHandle ArrayHandleType;

  MinMaxPairType initialValue =
    make_Pair(ResultType(vtkm::Infinity64()),
              ResultType(vtkm::NegativeInfinity64()));

  vtkm::cont::ArrayHandleTransform<MinMaxPairType, ArrayHandleType,
    internal::InputToOutputTypeTransform<3> > input(this->GetContour(contour)
                                                    .GetVertices());

  MinMaxPairType result = Algorithm::Reduce(input, initialValue,
                                            internal::MinMax<3>());

  for (unsigned i=0;i<3;i++)
    {
    bounds[2*i] = result.first[i];
    bounds[2*i+1] = result.second[i];
    }
}

//----------------------------------------------------------------------------
void PyFRContourData::ComputeBounds(FPType* bounds) const
{
  for (unsigned i=0;i<3;i++)
    {
    bounds[2*i] = std::numeric_limits<FPType>::max();
    bounds[2*i+1] = std::numeric_limits<FPType>::min();
    }
  for (unsigned i=0;i<this->GetNumberOfContours();i++)
    {
    FPType b[6];
    this->ComputeContourBounds(i,b);
    for (unsigned j=0;j<3;j++)
      {
      int jj = 2*j;
      bounds[jj] = (bounds[jj] < b[jj] ? bounds[jj] : b[jj]);
      jj++;
      bounds[jj] = (bounds[jj] > b[jj] ? bounds[jj] : b[jj]);
      }
    }
}

//----------------------------------------------------------------------------
void PyFRContourData::SetColorRange(FPType min,FPType max)
{
  for (std::vector<PyFRContour>::iterator it=this->Contours.begin();
  it!=this->Contours.end();++it)
    {
    (*it).GetColorTable().SetRange(min,max);
    }
}

namespace transfer
{

typedef ::vtkm::cont::DeviceAdapterTagCuda CudaTag;

//----------------------------------------------------------------------------
template<typename HandleType>
void to_gl(vtkm::Vec<vtkm::Float64,3>, const HandleType& handle, unsigned int& glHandle)
{
  //make an implicit wrapper to float32 around the float64 array
  vtkm::cont::ArrayHandleCast<vtkm::Vec<vtkm::Float32,3>,HandleType> asF32 =
    vtkm::cont::make_ArrayHandleCast(handle, vtkm::Vec<vtkm::Float32,3>());

  //transfer the array to openGL now as a float32 array
  vtkm::opengl::TransferToOpenGL(asF32, glHandle, CudaTag());
}

//----------------------------------------------------------------------------
template<typename HandleType>
void to_gl(vtkm::Float32, const HandleType& handle, unsigned int& glHandle)
{
  vtkm::opengl::TransferToOpenGL(handle, glHandle, CudaTag());
}

//----------------------------------------------------------------------------
void coords(PyFRContourData* data, int index, unsigned int& glHandle)
{
  to_gl(FPType(), data->GetContour(index).GetVertices(), glHandle);
}

//----------------------------------------------------------------------------
void normals(PyFRContourData* data, int index, unsigned int& glHandle)
{
  to_gl(FPType(), data->GetContour(index).GetNormals(), glHandle);
}

//----------------------------------------------------------------------------
void colors(PyFRContourData* data, int index, unsigned int& glHandle)
{
  //no need to worry about conversion, since this is always Vec4 of uint8's
  vtkm::opengl::TransferToOpenGL( data->GetContour(index).GetColorData(),
                                  glHandle,
                                  CudaTag());
}

} //namespace transfer
